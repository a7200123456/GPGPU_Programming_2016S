#include "hip/hip_runtime.h"
#include "lab2.h"
#define TIMESTEP 0.04
#define DIFF 0.0001
#define VISC 0.000015
static const unsigned NFRAME = 240;
static const unsigned W = 640;
static const unsigned H = 480;

void SWAP(float* A, float* B) {
    float *d_temp;
    
    hipMalloc((void **) &d_temp      , W*H*sizeof(float)); 
   
    hipMemcpy(d_temp, A     , W*H*sizeof(float),hipMemcpyDeviceToDevice); 
    hipMemcpy(A     , B     , W*H*sizeof(float),hipMemcpyDeviceToDevice); 
    hipMemcpy(B     , d_temp, W*H*sizeof(float),hipMemcpyDeviceToDevice); 
  
    hipFree(d_temp); 
}


float Lab2VideoGenerator::h_dens[640*480] = {};
struct Lab2VideoGenerator::Impl {
	int t = 0;
};

Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};

__global__ void init_dens(float* d_dens , float* d_dens_old,int t) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
  if (idx>=(W/2-100) && idx<(W/2+100) && idy>=(H/2-100) && idy<(H/2+100)){
    if(t==0)
      d_dens[idy*W+idx] = 64;
    else
      d_dens[idy*W+idx] = d_dens_old[idy*W+idx];    
  }
  else{
    if(t==0)
      d_dens[idy*W+idx] = 0;
    else
      d_dens[idy*W+idx] = d_dens_old[idy*W+idx];    
  }
}

__global__ void add_source(float* d_dens,float* d_dens_old ,float dt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
      d_dens[idy*W+idx] += (d_dens_old[idy*W+idx]*dt) ;
}

__global__ void diff_dens(float* d_dens,float* d_dens_old , float diff, float dt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int a = dt*diff*W*H;
  
    if (idx>=1 && idx<(W-1) && idy>=1 && idy<(H-1)){
      for(int k=0; k < 20; k++){
        d_dens[idy*W+idx] = (d_dens_old[idy*W+idx] + a*(d_dens[(idy-1)*W+idx] + d_dens[(idy+1)*W+idx] + d_dens[idy*W+(idx-1)] + d_dens[idy*W+(idx+1)]  ))/(1+4*a);
      }
    }
}

__global__ void output_yuv(uint8_t* yuv , float* result, int t) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(result[idx] >=255)
      yuv[idx] =  255;
    else if (result[idx] < 0)
      yuv[idx] = 0;
    else
      yuv[idx] = result[idx];
}

void Lab2VideoGenerator::Generate(uint8_t *yuv) {
  
  float *d_dens;
  float *d_dens_old;
  float *d_vel_x,*d_vel_y,*d_vel_x_old,*d_vel_y_old;
  
  hipMalloc((void **) &d_dens      , W*H*sizeof(float)); 
  hipMalloc((void **) &d_dens_old  , W*H*sizeof(float)); 
  //hipMalloc((void **) &d_vel_x     , W*H*sizeof(float)); 
  //hipMalloc((void **) &d_vel_x_old , W*H*sizeof(float)); 
  //hipMalloc((void **) &d_vel_y     , W*H*sizeof(float)); 
  //hipMalloc((void **) &d_vel_y_old , W*H*sizeof(float)); 
  
  hipMemcpy(d_dens_old, h_dens, W*H*sizeof(float),hipMemcpyHostToDevice); 
  
  dim3 blocks(W/16, H/16);
  dim3 threads(16, 16);
  
  init_dens<<<blocks, threads>>>(d_dens, d_dens_old, impl->t);
  //add_source<<<blocks, threads>>>(d_dens,d_dens_old,TIMESTEP);
  SWAP(d_dens, d_dens_old);
  diff_dens<<<blocks, threads>>>(d_dens,d_dens_old,DIFF,TIMESTEP);
  
  output_yuv<<<W*H/512, 512>>>(yuv, d_dens,impl->t);
  
  hipMemcpy(h_dens, d_dens, W*H*sizeof(float),hipMemcpyDeviceToHost); 
  
  //hipMemset(yuv, (impl->t)*255/NFRAME, W*H);
	hipMemset(yuv+W*H, 128, W*H/2);
  hipFree(d_dens); 
  hipFree(d_dens_old ); 
  //hipFree(d_vel_x    ); 
  //hipFree(d_vel_x_old); 
  //hipFree(d_vel_y    ); 
  //hipFree(d_vel_y_old); 
  
	++(impl->t);
}
